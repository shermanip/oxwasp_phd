#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

//See empiricalNullFilter - this is the main entry
//Notes: row major
__constant__ int roiWidth;
__constant__ int roiHeight;
__constant__ int cacheWidth;
__constant__ int cacheHeight;
__constant__ int kernelRadius;
__constant__ int kernelHeight;
__constant__ int nPoints; //number of points in kernel
__constant__ int nInitial; //number of initial values for Newton-Raphson
__constant__ int nStep; //number of steps for Newton-Raphson
__constant__ int cacheSharedWidth; //the width of the shared memory cache

/**FUNCTION: Get derivative
 * Set dxLnF to contain derivatives of the density estimate (of values in the
 *     kernel) evaluated at a point
 * PARAMETERS:
 *   cache: see empiricalNullFilter
 *   bandwidth: see findMode
 *   kernelPointers: see empiricalNullFilter
 *   value: where to evaluate the density estimate and the derivatives
 *   dxLnF: MODIFIED 3-element array, to store results. The elements are:
 *     1. the density (ignore any constant multiplied to it) (NOT THE LOG)
 *     2. the first derivative of the log density
 *     3. the second derivative of the log density
 */
__device__ void getDLnDensity(float* cacheShared, float bandwidth,
    int* kernelPointers, float* value, float* dxLnF) {

  //coordinates of the centre of the kernel
  int x0 = threadIdx.x;
  int y0 = threadIdx.y;

  //variables when going through all pixels in the kernel
  int cachePointer; //pointer for cache
  float z; //value of a pixel when looping through kernel
  float y = y0 - kernelRadius; //y coordinate when looping through kernel
  float sumKernel[3] = {0.0f}; //store sums of weights
  float phiZ; //weight, use Gaussian kernel

  //for each row in the kernel
  for (int i=0; i<2*kernelHeight; i++) {
    //for each column for this row
    for (int dx=kernelPointers[i++]; dx<=kernelPointers[i]; dx++) {
      //append to sum
      cachePointer = (y+kernelRadius)*cacheSharedWidth + x0 + dx + kernelRadius;
      z = (cacheShared[cachePointer] - *value) / bandwidth;
      phiZ = expf(-z*z/2);
      sumKernel[0] += phiZ;
      sumKernel[1] += phiZ * z;
      sumKernel[2] += phiZ * z * z;
    }
    y++;
  }

  //work out derivatives
  float normaliser = bandwidth*sumKernel[0];
  dxLnF[0] = sumKernel[0];
  dxLnF[1] = sumKernel[1] / normaliser;
  dxLnF[2] = (sumKernel[0]*(sumKernel[2] - sumKernel[0])
      - sumKernel[1]*sumKernel[1]) / (normaliser * normaliser);
}

/**FUNCTION: Find mode
 * Use Newton-Raphson to find the maximum value of the density estimate. Uses
 *     the passed nullMean as the initial value and modifies it at each step,
 *     ending up with a final answer.
 * The second derivative of the log density and the density (up to a constant)
 *     at the final answer is stored in secondDiffLn and densityAtMode.
 * PARAMETERS:
 *   cache: see empiricalNullFilter
 *   bandwidth: bandwidth for the density estimate
 *   kernelPointers: see empiricalNullFilter
 *   nullMean: MODIFIED initial value for the Newton-Raphson method, modified
 *       to contain the final answer
 *   secondDiffLn: MODIFIED second derivative of the log density
 * RETURNS: true if sucessful, false otherwise
 */
__device__ bool findMode(float* cacheShared, float bandwidth, int* kernelPointers,
    float* nullMean, float* secondDiffLn, float* densityAtMode) {
  float dxLnF[3];
  //nStep of Newton-Raphson
  for (int i=0; i<nStep; i++) {
    getDLnDensity(cacheShared, bandwidth, kernelPointers, nullMean, dxLnF);
    *nullMean -= dxLnF[1] / dxLnF[2];
  }
  getDLnDensity(cacheShared, bandwidth, kernelPointers, nullMean, dxLnF);
  //need to check if answer is valid
  if (isfinite(*nullMean) && isfinite(dxLnF[0]) && isfinite(dxLnF[1])
      && isfinite(dxLnF[2]) && (dxLnF[2] < 0)) {
    *densityAtMode = dxLnF[0];
    *secondDiffLn = dxLnF[2];
    return true;
  } else {
    return false;
  }
}

/**KERNEL: Empirical Null Filter
 * Does the empirical null filter on the pixels in cache, giving the empirical
 *     null mean (aka mode) and the empirical null std.
 * PARAMETERS:
 *   cache: array of pixels in the cache
 *   initialSigmaRoi: array of pixels (same size as the ROI) containing standard
 *       deviations, used for producing random initial values for Newton-Raphson
 *   bandwidthRoi: array of pixels (same size as the ROI) containing
 *   kernelPointers: array (even number of elements, size 2*kernelHeight)
 *       containing pairs of integers, indicates for each row the position from
 *       the centre of the kernel
 *   nullMeanRoi: MODIFIED array of pixels (same size as ROI), pass results of
 *       median filter here to be used as initial values. Modified to contain
 *       the empricial null mean afterwards.
 *   nullStdRoi: MODIFIED array of pixels (same size as ROI) to contain the
 *       empirical null std
 */
extern "C" __global__ void empiricalNullFilter(float* cache,
    float* initialSigmaRoi, float* bandwidthRoi, int* kernelPointers,
    float* nullMeanRoi, float* nullStdRoi) {

  int x0 = threadIdx.x + blockIdx.x * blockDim.x;
  int y0 = threadIdx.y + blockIdx.y * blockDim.y;
  int threadId = y0*roiWidth + x0;

  extern __shared__ float cacheShared[];

  if (x0 < roiWidth && y0 < roiHeight) {

    //variables when going through all pixels in the kernel
    int cachePointer; //pointer for cache
    int cacheSharedPointer; //pointer for cacheShared
    float y = y0 - kernelRadius; //y coordinate when looping through kernel

    //for each row in the kernel
    for (int i=0; i<2*kernelHeight; i++) {
      //for each column for this row
      for (int dx=kernelPointers[i++]; dx<=kernelPointers[i]; dx++) {
        //append to sum
        cachePointer = (y+kernelRadius)*cacheWidth + x0 + dx + kernelRadius;
        cacheSharedPointer = (threadIdx.y+kernelRadius+y-y0)*cacheSharedWidth
            + threadIdx.x + dx + kernelRadius;
        cacheShared[cacheSharedPointer] = cache[cachePointer];
      }
      y++;
    }

    //for rng
    hiprandState_t state;
    hiprand_init(0, threadId, 0, &state);

    int roiPointer = y0*roiWidth + x0;

    //try different initial values, the first one is the median, then add normal
        //noise to the median for different initial values
    float initial0 = nullMeanRoi[roiPointer]; //median
    float nullMean = initial0; //store the locatio of the mode
    float sigma = initialSigmaRoi[roiPointer]; //how much noise to add

    float bandwidth = bandwidthRoi[roiPointer]; //bandwidth for density estimate
    bool isSuccess; //indiciate if newton-raphson was sucessful
    float densityAtMode; //density for this particular mode
    //second derivative of the log density, to set empirical null std
    float secondDiffLn;

    //keep solution with the highest density
    float maxDensityAtMode = -INFINITY;

    for (int i=0; i<nInitial; i++) {
      isSuccess = findMode(cacheShared, bandwidth, kernelPointers, &nullMean,
          &secondDiffLn, &densityAtMode);
      //keep nullMean and nullStd with the highest density
      if (isSuccess) {
        if (densityAtMode > maxDensityAtMode) {
          maxDensityAtMode = densityAtMode;
          nullMeanRoi[roiPointer] = nullMean;
          nullStdRoi[roiPointer] = powf(-secondDiffLn, -0.5f);
        }
      }

      //try different initial value
      nullMean = initial0 + sigma * hiprand_normal(&state);
    }

  }
}
